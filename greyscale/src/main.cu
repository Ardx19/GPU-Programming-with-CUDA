#include <iostream>
#include <string>
#include <vector>
#include <dirent.h> // For directory reading

// Define STB_IMAGE_IMPLEMENTATION and STB_IMAGE_WRITE_IMPLEMENTATION
// in exactly one C or C++ file to create the implementation.
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include "grayscale.cuh"

// Helper function to check for CUDA errors
void checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(result));
        exit(1);
    }
}

int main() {
    const std::string input_dir = "input_data";
    const std::string output_dir = "output_data";

    DIR *dir;
    struct dirent *ent;
    
    if ((dir = opendir(input_dir.c_str())) != NULL) {
        std::cout << "Starting batch image processing..." << std::endl;
        
        // Iterate over all the files in the input directory
        while ((ent = readdir(dir)) != NULL) {
            std::string filename = ent->d_name;
            if (filename == "." || filename == "..") {
                continue;
            }

            std::string input_path = input_dir + "/" + filename;
            
            // 1. Load image from disk using stb_image
            int width, height, channels;
            unsigned char *h_input_img = stbi_load(input_path.c_str(), &width, &height, &channels, 0);

            if (h_input_img == NULL) {
                std::cerr << "Error loading image: " << input_path << std::endl;
                continue;
            }

            // We only process RGB or RGBA images
            if (channels < 3) {
                std::cout << "Skipping non-color image: " << filename << std::endl;
                stbi_image_free(h_input_img);
                continue;
            }

            std::cout << "Processing: " << filename << " (" << width << "x" << height << ")" << std::endl;

            // 2. Allocate memory on the host and device
            size_t img_size = width * height * channels * sizeof(unsigned char);
            size_t gray_img_size = width * height * sizeof(unsigned char);

            unsigned char *h_output_gray_img = (unsigned char*)malloc(gray_img_size);
            unsigned char *d_input_img, *d_output_gray_img;

            checkCuda(hipMalloc((void**)&d_input_img, img_size));
            checkCuda(hipMalloc((void**)&d_output_gray_img, gray_img_size));

            // 3. Copy image data from host to device
            checkCuda(hipMemcpy(d_input_img, h_input_img, img_size, hipMemcpyHostToDevice));

            // 4. Launch the CUDA kernel
            convertToGrayscale(d_input_img, d_output_gray_img, width, height, channels);
            
            // Check for any errors during kernel execution
            checkCuda(hipGetLastError());
            checkCuda(hipDeviceSynchronize());

            // 5. Copy the result back from device to host
            checkCuda(hipMemcpy(h_output_gray_img, d_output_gray_img, gray_img_size, hipMemcpyDeviceToHost));
            
            // 6. Save the grayscale image to disk
            std::string output_path = output_dir + "/gray_" + filename;
            // Save as PNG to avoid compression artifacts and handle single channel easily
            if (output_path.find(".jpg") != std::string::npos) {
                 output_path.replace(output_path.find(".jpg"), 4, ".png");
            }
            stbi_write_png(output_path.c_str(), width, height, 1, h_output_gray_img, width * sizeof(unsigned char));

            // 7. Free memory
            stbi_image_free(h_input_img);
            free(h_output_gray_img);
            hipFree(d_input_img);
            hipFree(d_output_gray_img);
        }
        closedir(dir);
        std::cout << "Processing complete. Results are in the 'output_data' directory." << std::endl;
    } else {
        std::cerr << "Error: Could not open input directory '" << input_dir << "'" << std::endl;
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}
